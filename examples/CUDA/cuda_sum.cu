#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	auto step_size = 1;
	int number_of_threads = blockDim.x;
	
	printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
		}

		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}

int main()
{
	const auto count = 16;
	const int size = count * sizeof(int);
	//int h[] = {13, 27, 15, 14, 33, 2, 24, 6};
	
	
	int h[] = {13, 27, 15, 14, 33, 2, 24, 6, 1, 120, 40, 50 , 60, 12, 13, 18};
	
	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);
	delete[] h;

	return 0;
}